#include <vector>
#include <iostream>

#include <amgcl/backend/cuda.hpp>
#include <amgcl/adapter/crs_tuple.hpp>
#include <amgcl/make_solver.hpp>
#include <amgcl/amg.hpp>
#include <amgcl/coarsening/smoothed_aggregation.hpp>
#include <amgcl/relaxation/spai0.hpp>
#include <amgcl/solver/bicgstab.hpp>

#include <amgcl/io/mm.hpp>
#include <amgcl/profiler.hpp>

int main(int argc, char *argv[]) {
    // The matrix and the RHS file names should be in the command line options:
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <matrix.mtx> <rhs.mtx>" << std::endl;
        return 1;
    }

    // Show the name of the GPU we are using:
    int device;
    hipDeviceProp_t prop;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    std::cout << prop.name << std::endl;

    // The profiler:
    amgcl::profiler<> prof("poisson3Db");

    // Read the system matrix and the RHS:
    ptrdiff_t rows, cols;
    std::vector<ptrdiff_t> ptr, col;
    std::vector<double> val, rhs;

    prof.tic("read");
    std::tie(rows, cols) = amgcl::io::mm_reader(argv[1])(ptr, col, val);
    std::cout << "Matrix " << argv[1] << ": " << rows << "x" << cols << std::endl;

    std::tie(rows, cols) = amgcl::io::mm_reader(argv[2])(rhs);
    std::cout << "RHS " << argv[2] << ": " << rows << "x" << cols << std::endl;
    prof.toc("read");

    // We use the tuple of CRS arrays to represent the system matrix.
    // Note that std::tie creates a tuple of references, so no data is actually
    // copied here:
    auto A = std::tie(rows, ptr, col, val);

    // Compose the solver type
    typedef amgcl::backend::cuda<double> Backend;
    typedef amgcl::make_solver<
        amgcl::amg<
            Backend,
            amgcl::coarsening::smoothed_aggregation,
            amgcl::relaxation::spai0
            >,
        amgcl::solver::bicgstab<Backend>
        > Solver;

    // We need to initialize the CUSPARSE library and pass the handle to AMGCL
    // in backend parameters:
    Backend::params bprm;
    hipsparseCreate(&bprm.cusparse_handle);

    // There is no way to pass the backend parameters without passing the
    // solver parameters, so we also need to create those. But we can leave
    // them with the default values:
    Solver::params prm;

    // Initialize the solver with the system matrix:
    prof.tic("setup");
    Solver solve(A, prm, bprm);
    prof.toc("setup");

    // Show the mini-report on the constructed solver:
    std::cout << solve << std::endl;

    // Solve the system with the zero initial approximation.
    // The RHS and the solution vectors should reside in the GPU memory:
    int iters;
    double error;
    thrust::device_vector<double> f(rhs);
    thrust::device_vector<double> x(rows, 0.0);

    prof.tic("solve");
    std::tie(iters, error) = solve(f, x);
    prof.toc("solve");

    // Output the number of iterations, the relative error,
    // and the profiling data:
    std::cout << "Iters: " << iters << std::endl
              << "Error: " << error << std::endl
              << prof << std::endl;
}
